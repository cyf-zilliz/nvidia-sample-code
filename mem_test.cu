#include <stdio.h>
#include <thread>
#include <unistd.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#define GRID_X  128
#define BLOCK_X 128

__global__ void proc(int *mem){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while(true){
        mem[idx]++;
    }
}


void run_thread()
{
    hipSetDevice(0);
    //cudaStream_t s1;
    //cudaStreamCreate(&s1);
    int *pdata;
    hipMallocManaged(&pdata,GRID_X*BLOCK_X);
    memset(pdata,0,sizeof(int)*GRID_X*BLOCK_X);
    sleep(1);
    std::cout << "start kernel in run_thread" << std::endl;
    //proc<<<GRID_X,BLOCK_X,0,s1>>>(pdata);
    //cudaStreamSynchronize(s1);
    proc<<<GRID_X,BLOCK_X>>>(pdata);
    hipDeviceSynchronize();
    std::cout << "end kernel in run_thread" << std::endl;
}

void free_thread()
{
    hipSetDevice(1);
    int *pdata;
    hipMallocManaged(&pdata,GRID_X*BLOCK_X);
    sleep(5);
    std::cout << "start hipFree in free_thread" << std::endl;
    hipFree(pdata);
    std::cout << "end hipFree in free_thread" << std::endl;
}


int main(){

    std::thread th0(&run_thread);
    std::thread th1(&free_thread);
    th0.join();
    th1.join();
    return 0;
}

